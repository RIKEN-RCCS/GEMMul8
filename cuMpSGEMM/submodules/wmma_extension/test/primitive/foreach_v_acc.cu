#include "hip/hip_runtime.h"
#include <iostream>
#include <type_traits>
#include <wmma_extension/wmma_extension.hpp>

#ifndef TEST_ARCH
#define TEST_ARCH (-1)
#endif

//#define TEST_TF32

#ifndef TEST_TF32
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 16;
#else
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 8;
#endif

template <class T, class S>
__device__ __host__ T convert(const S);
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

__global__ void test_foreach_v_acc_kernel(
		float* const dst,
		const float* const src,
		const nvcuda::wmma::layout_t layout
		) {
	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, M, N, K, float> frag_c;
	nvcuda::wmma::load_matrix_sync(frag_c, src, M, layout);
	mtk::wmma::foreach_v<decltype(frag_c)>(layout, [&](const unsigned* frag_index_list, const unsigned fragment_index_count, const unsigned mem_index) {
				for (unsigned i = 0; i < fragment_index_count; i++) {
					dst[mem_index] = frag_c.x[frag_index_list[i]];
				}
			});
}

void test(const nvcuda::wmma::layout_t layout) {
	std::printf("-- test (%s) --\n", __FILE__);
	std::printf("arch   : %d\n", TEST_ARCH);
	if (layout == nvcuda::wmma::mem_col_major) {
		std::printf("layout : col_major\n");
	} else {
		std::printf("layout : row_major\n");
	}
	std::printf("size   : %lu, %lu, %lu\n", M, N, K);
	float* src_mem;
	float* dst_mem;

	hipHostMalloc(&src_mem, M * N * sizeof(float));
	hipHostMalloc(&dst_mem, M * sizeof(float));

	for (std::size_t i = 0; i < M * N; i++) {
		src_mem[i] = static_cast<float>(i);
	}

	hipDeviceSynchronize();
	test_foreach_v_acc_kernel<<<1, 32>>>(dst_mem, src_mem, layout);
	hipDeviceSynchronize();

	float error = 0.f;
	for (std::size_t i = 0; i < M; i++) {
		error = std::max(std::abs(dst_mem[i] - src_mem[i]), error);
	}
	std::printf("error = %e\n", error);
}

int main() {
	test(nvcuda::wmma::mem_row_major);
	test(nvcuda::wmma::mem_col_major);
	test(nvcuda::wmma::mem_row_major);
	test(nvcuda::wmma::mem_col_major);
}
