#include "hip/hip_runtime.h"
#include <iostream>
#include <type_traits>
#include <wmma_extension/wmma_extension.hpp>

#ifndef TEST_ARCH
#define TEST_ARCH (-1)
#endif

//#define TEST_TF32

#ifndef TEST_TF32
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 16;
using ab_type = half;
#else
constexpr std::size_t M = 16;
constexpr std::size_t N = 16;
constexpr std::size_t K = 8;
using ab_type = nvcuda::wmma::precision::tf32;
#endif

using storage_t = typename mtk::wmma::detail::common::storage_t<ab_type>::type;

template <class T, class S>
__device__ __host__ typename mtk::wmma::detail::common::storage_t<T>::type convert(const S);
template <> __device__ __host__ typename mtk::wmma::detail::common::storage_t<float>::type convert<float, float>(const float a) {return a;}
template <> __device__ __host__ typename mtk::wmma::detail::common::storage_t<float>::type convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ typename mtk::wmma::detail::common::storage_t<half >::type convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ typename mtk::wmma::detail::common::storage_t<half >::type convert<half , half >(const half  a) {return a;}

template <class T>
__device__ T m_abs(const T a) {
	if (a >= convert<T, float>(0)) return a;
	return -a;
}

template <class Use, class layout>
__global__ void test_foreach_v_kernel(
		const storage_t* const src,
		const storage_t* const cor
		) {
	nvcuda::wmma::fragment<Use, M, N, K, ab_type, layout> vec_frag;
	mtk::wmma::fill_zero(vec_frag);
	mtk::wmma::foreach_v<decltype(vec_frag)>([&](const unsigned frag_index_list[], const unsigned frag_index_count, const unsigned mem_index) {
				for (unsigned i = 0; i < frag_index_count; i++) {
					vec_frag.x[frag_index_list[i]] = convert<storage_t>(src[mem_index]);
				}
			});

	nvcuda::wmma::fragment<Use, M, N, K, ab_type, layout> cor_frag;
	nvcuda::wmma::load_matrix_sync(cor_frag, cor, M);

	storage_t error = convert<storage_t, float>(0.0f);
	for (unsigned i = 0; i < vec_frag.num_elements; i++) {
		error += m_abs(vec_frag.x[i] - cor_frag.x[i]);
	}
	printf("[%2u] error = %e\n", threadIdx.x, convert<float>(error));
}

template <class Use, class layout>
void test() {
	std::printf("-- test (%s) --\n", __FILE__);
	std::size_t cor_size = 0;
	std::size_t vec_length = 0;
	std::printf("arch   : %d\n", TEST_ARCH);
	if (std::is_same<layout, nvcuda::wmma::col_major>::value) {
		std::printf("layout : col_major\n");
	} else {
		std::printf("layout : row_major\n");
	}
	if (std::is_same<float, ab_type>::value)
		std::printf("type   : float\n");
	if (std::is_same<half, ab_type>::value)
		std::printf("type   : half\n");
	if (std::is_same<nvcuda::wmma::precision::tf32, ab_type>::value)
		std::printf("type   : tf32\n");

	if (std::is_same<nvcuda::wmma::matrix_a, Use>::value) {
		std::printf("use    : a\n");
		cor_size = M * K;
		if (std::is_same<nvcuda::wmma::col_major, layout>::value) {
			vec_length = M;
		} else {
			vec_length = K;
		}
	}
	if (std::is_same<nvcuda::wmma::matrix_b, Use>::value) {
		std::printf("use    : b\n");
		cor_size = N * K;
		if (std::is_same<nvcuda::wmma::col_major, layout>::value) {
			vec_length = K;
		} else {
			vec_length = N;
		}
	}
	std::printf("size   : %lu, %lu, %lu\n", M, N, K);

	storage_t* src_mem;
	storage_t* cor_mem;

	hipHostMalloc(&src_mem, M * sizeof(storage_t));
	hipHostMalloc(&cor_mem, cor_size * sizeof(storage_t));

	for (std::size_t i = 0; i < cor_size; i++) {
		cor_mem[i] = convert<storage_t, float>(0);
	}

	for (std::size_t i = 0; i < vec_length; i++) {
		const float v = i / 3.0f;
		src_mem[i] = convert<storage_t, float>(v);
		cor_mem[i] = convert<storage_t, float>(v);
	}

	hipDeviceSynchronize();
	test_foreach_v_kernel<Use, layout><<<1, 32>>>(src_mem, cor_mem);
	hipDeviceSynchronize();
}

int main() {
	test<nvcuda::wmma::matrix_a, nvcuda::wmma::col_major>();
	test<nvcuda::wmma::matrix_a, nvcuda::wmma::row_major>();

	test<nvcuda::wmma::matrix_b, nvcuda::wmma::col_major>();
	test<nvcuda::wmma::matrix_b, nvcuda::wmma::row_major>();
}
