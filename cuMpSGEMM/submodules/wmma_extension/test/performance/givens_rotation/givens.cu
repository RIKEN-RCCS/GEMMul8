#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <wmma_extension/wmma_extension.hpp>

constexpr unsigned warp_size = 32;
constexpr unsigned block_size = 256;
constexpr unsigned test_count = 1u << 12;

template <unsigned DIM>
__device__ void cp_matrix(
		half2* const smem,
		const half2* const gmem
		) {
	for (unsigned i = 0; i < DIM * DIM / 2; i += warp_size * 4) {
		const unsigned index = i + (threadIdx.x & 0x1fu);
		reinterpret_cast<uint4*>(smem)[index] = reinterpret_cast<const uint4*>(gmem)[index];
	}
}

template <unsigned DIM, class GivensMatGen>
__global__ void batched_givens_kernel(
		half* const ptr,
		const unsigned batch_size) {
	__shared__ half smem[DIM * DIM * block_size / warp_size];

	const unsigned gi = 5;
	const unsigned gj = 6;
	const float theta = M_PI / 6;
	half* const smem_ptr = smem + DIM * DIM * (threadIdx.x / warp_size);

	const unsigned matrix_id = threadIdx.x + blockIdx.x * blockDim.x / warp_size;
	if (matrix_id >= batch_size) return;

	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_a;
	cp_matrix<DIM>(
			reinterpret_cast<half2*>(smem_ptr),
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size))
			);
	GivensMatGen{}(frag_a, gi, gj, theta, smem_ptr);
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_b;
	nvcuda::wmma::load_matrix_sync(frag_b, smem_ptr, DIM);

	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, DIM, DIM, DIM, half> frag_c;
	nvcuda::wmma::fill_fragment(frag_c, 0.f);

	nvcuda::wmma::mma_sync(frag_c, frag_a, frag_b, frag_c);

	nvcuda::wmma::store_matrix_sync(smem_ptr, frag_c, DIM, nvcuda::wmma::mem_col_major);
	cp_matrix<DIM>(
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size)),
			reinterpret_cast<half2*>(smem_ptr)
			);
}

template <unsigned DIM, class GivensMatGen>
__global__ void batched_givens_kernel(
		half* const ptr,
		const unsigned gi, const unsigned gj,
		const unsigned batch_size) {
	__shared__ half smem[DIM * DIM * block_size / warp_size];

	const float theta = M_PI / 6;
	half* const smem_ptr = smem + DIM * DIM * (threadIdx.x / warp_size);

	const unsigned matrix_id = threadIdx.x + blockIdx.x * blockDim.x / warp_size;
	if (matrix_id >= batch_size) return;

	cp_matrix<DIM>(
			reinterpret_cast<half2*>(smem_ptr),
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size))
			);
	nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_b;
	nvcuda::wmma::load_matrix_sync(frag_b, smem_ptr, DIM);

	nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major> frag_a;
	GivensMatGen{}(frag_a, gi, gj, theta, smem_ptr);

	nvcuda::wmma::fragment<nvcuda::wmma::accumulator, DIM, DIM, DIM, half> frag_c;
	nvcuda::wmma::fill_fragment(frag_c, 0.f);

	nvcuda::wmma::mma_sync(frag_c, frag_a, frag_b, frag_c);

	nvcuda::wmma::store_matrix_sync(smem_ptr, frag_c, DIM, nvcuda::wmma::mem_col_major);
	cp_matrix<DIM>(
			reinterpret_cast<half2*>(ptr + DIM * DIM * ((threadIdx.x + block_size / warp_size * blockIdx.x) / warp_size)),
			reinterpret_cast<half2*>(smem_ptr)
			);
}

template <unsigned DIM>
struct GivensMatGenWMMA {
	__device__ GivensMatGenWMMA(){}
	__device__ void operator()(
			nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major>& frag,
			const unsigned gi,
			const unsigned gj,
			const float theta,
			half* const smem
			) const {
		half2* smem_h2_ptr = reinterpret_cast<half2*>(smem);
		for (unsigned i = 0; i < DIM * DIM / 2; i += warp_size) {
			const unsigned index = i + (threadIdx.x & 0x1fu);
			*reinterpret_cast<uint32_t*>(&(smem_h2_ptr[index])) = 0u;
		}
		__syncwarp();
		const auto lane_id = (threadIdx.x & 0x1f);
		if (lane_id < 16) {
			smem[(1 + DIM) * lane_id] = __float2half(1.f);
		}
		__syncwarp();
		if (lane_id == 0) {
			const auto c = __float2half(__cosf(theta));
			smem[gi + gi * DIM] = c;
			smem[gj + gj * DIM] = c;
			const auto s = __float2half(__sinf(theta));
			smem[gi + gj * DIM] =-s;
			smem[gj + gi * DIM] = s;
		}
		__syncwarp();
		nvcuda::wmma::load_matrix_sync(frag, smem, DIM);
	}
};

template <unsigned DIM>
struct GivensMatGenWMMAe {
	__device__ GivensMatGenWMMAe(){}
	__device__ void operator()(
			nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, DIM, DIM, DIM, half, nvcuda::wmma::col_major>& frag,
			const unsigned gi,
			const unsigned gj,
			const float theta,
			half* const
			) const {
		mtk::wmma::fill_zero(frag);
		unsigned tid_list[2];
		unsigned fid_list[2];
		unsigned list_size;
		for (unsigned i = 0; i < DIM; i++) {
			mtk::wmma::map<decltype(frag)>(tid_list, fid_list, list_size, i, i);
			for (unsigned k = 0; k < list_size; k++) {
				if (threadIdx.x == tid_list[k]) {
					frag.x[fid_list[k]] = 1.0f;
				}
			}
		}
		const auto c = __float2half(__cosf(theta));
		mtk::wmma::map<decltype(frag)>(tid_list, fid_list, list_size, gi, gi);
		for (unsigned k = 0; k < list_size; k++) {
			if (threadIdx.x == tid_list[k]) {
				frag.x[fid_list[k]] = c;
			}
		}
		mtk::wmma::map<decltype(frag)>(tid_list, fid_list, list_size, gj, gj);
		for (unsigned k = 0; k < list_size; k++) {
			if (threadIdx.x == tid_list[k]) {
				frag.x[fid_list[k]] = c;
			}
		}
		const auto s = __float2half(__sinf(theta));
		mtk::wmma::map<decltype(frag)>(tid_list, fid_list, list_size, gi, gj);
		for (unsigned k = 0; k < list_size; k++) {
			if (threadIdx.x == tid_list[k]) {
				frag.x[fid_list[k]] = -s;
			}
		}
		mtk::wmma::map<decltype(frag)>(tid_list, fid_list, list_size, gj, gi);
		for (unsigned k = 0; k < list_size; k++) {
			if (threadIdx.x == tid_list[k]) {
				frag.x[fid_list[k]] = s;
			}
		}
	}
};

template <class T>
std::string get_class_name();
template <> std::string get_class_name<GivensMatGenWMMA <16>>() {return "wmma_16";}
template <> std::string get_class_name<GivensMatGenWMMAe<16>>() {return "wmmae_16";}

template <unsigned DIM, class GivensMatGen>
void batched_givens(
		half* const ptr,
		const unsigned batch_size
		) {
	const unsigned grid_size = (batch_size * warp_size + block_size - 1) / block_size;
	batched_givens_kernel<DIM, GivensMatGen><<<grid_size, block_size>>>(ptr, batch_size);
}

template <unsigned DIM, class GivensMatGen>
void test_batched_kernel(
		const unsigned batch_size
		) {
	half* input_matrix;
	hipMalloc(&input_matrix, sizeof(half) * DIM * DIM * batch_size);
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned c = 0; c < test_count; c++) {
		batched_givens<DIM, GivensMatGen>(
				input_matrix,
				batch_size);
	}
	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();
	hipFree(input_matrix);

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() / static_cast<double>(test_count) * 1e-6;

	std::printf("%u,1,%s,%e\n", batch_size, get_class_name<GivensMatGen>().c_str(), elapsed_time);
}

template <unsigned DIM, class GivensMatGen>
void batched_givens(
		half* const ptr,
		const unsigned gi, const unsigned gj,
		const unsigned batch_size
		) {
	const unsigned grid_size = (batch_size * warp_size + block_size - 1) / block_size;
	batched_givens_kernel<DIM, GivensMatGen><<<grid_size, block_size>>>(ptr, gi, gj, batch_size);
}

template <unsigned DIM, class GivensMatGen>
void test_batched_kernel(
		const unsigned gi, const unsigned gj,
		const unsigned batch_size
		) {
	half* input_matrix;
	hipMalloc(&input_matrix, sizeof(half) * DIM * DIM * batch_size);
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned c = 0; c < test_count; c++) {
		batched_givens<DIM, GivensMatGen>(
				input_matrix,
				gi, gj,
				batch_size);
	}
	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();
	hipFree(input_matrix);

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() / static_cast<double>(test_count) * 1e-6;

	std::printf("%u,0,%s,%e\n", batch_size, get_class_name<GivensMatGen>().c_str(), elapsed_time);
}

int main() {
	std::printf("batch_size,embedded,api,time\n");
	for (unsigned i = 13; i <= 23; i++) {
		test_batched_kernel<16, GivensMatGenWMMA <16>>(1u << i);
		test_batched_kernel<16, GivensMatGenWMMAe<16>>(1u << i);
		test_batched_kernel<16, GivensMatGenWMMA <16>>(5, 6, 1u << i);
		test_batched_kernel<16, GivensMatGenWMMAe<16>>(5, 6, 1u << i);
	}
}
