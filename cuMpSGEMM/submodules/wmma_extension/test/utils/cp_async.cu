#include "hip/hip_runtime.h"
#include <iostream>
#include <wmma_extension/utils.hpp>

template <class T>
__host__ __device__ constexpr unsigned get_size_in_byte();
template <> __host__ __device__ constexpr unsigned get_size_in_byte<float >() {return 4;};
template <> __host__ __device__ constexpr unsigned get_size_in_byte<float2>() {return 8;};
template <> __host__ __device__ constexpr unsigned get_size_in_byte<float4>() {return 16;};

template <class T, unsigned block_size>
__global__ void cp_async_test_kernel(
		T* const dst_ptr,
		const T* const src_ptr
		) {
	__shared__ T smem[block_size];

	mtk::wmma::utils::cp_async::cp_async<get_size_in_byte<T>()>(smem + threadIdx.x, src_ptr + threadIdx.x);
	mtk::wmma::utils::cp_async::commit();

	mtk::wmma::utils::cp_async::wait_all();
	dst_ptr[threadIdx.x] = smem[threadIdx.x];
}

template <class T, unsigned block_size>
void cp_async_test() {
	T* d_input;
	T* d_output;
	T* h_input;
	T* h_output;

	hipMalloc(&d_input, sizeof(T) * block_size);
	hipMalloc(&d_output, sizeof(T) * block_size);
	hipHostMalloc(&h_input, sizeof(T) * block_size);
	hipHostMalloc(&h_output, sizeof(T) * block_size);

	for (unsigned i = 0; i < block_size * get_size_in_byte<T>() / 4; i++) {
		reinterpret_cast<float*>(h_input)[i] = i;
	}

	hipMemcpy(d_input, h_input, block_size * sizeof(T), hipMemcpyDefault);

	cp_async_test_kernel<T, block_size><<<1, block_size>>>(d_output, d_input);

	hipMemcpy(h_output, d_output, block_size * sizeof(T), hipMemcpyDefault);

	double max_error = 0;
	for (unsigned i = 0; i < block_size * get_size_in_byte<T>() / 4; i++) {
		const double diff = reinterpret_cast<float*>(h_output)[i] - reinterpret_cast<float*>(h_input)[i];
		max_error = std::max(std::abs(diff), max_error);
	}

	std::printf("%s[%2u Byte] error = %e\n", __func__, get_size_in_byte<T>(), max_error);

	hipFree(d_input);
	hipFree(d_output);
	hipHostFree(h_input);
	hipHostFree(h_output);
}

int main() {
	cp_async_test<float , 128>();
	cp_async_test<float2, 128>();
	cp_async_test<float4, 128>();
}

