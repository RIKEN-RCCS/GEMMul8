#include "hip/hip_runtime.h"
#include "gemmul8.hpp"
#include "matrixmarketio.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <dlfcn.h>

#define CHECK_CUDA(call)                                               \
    do {                                                               \
        hipError_t err = call;                                        \
        if (err != hipSuccess) {                                      \
            std::cerr << "CUDA error: " << hipGetErrorString(err)     \
                      << " (" << __FILE__ << ":" << __LINE__ << ")\n"; \
            exit(EXIT_FAILURE);                                        \
        }                                                              \
    } while (0)

#define CHECK_CUBLAS(call)                                                          \
    do {                                                                            \
        hipblasStatus_t status = call;                                               \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                      \
            std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << "\n"; \
            exit(EXIT_FAILURE);                                                     \
        }                                                                           \
    } while (0)

void fill_random(double *data, int n) {
    for (int i = 0; i < n; ++i) data[i] = static_cast<double>(rand()) / RAND_MAX;
}
void fill_random(float *data, int n) {
    for (int i = 0; i < n; ++i) data[i] = static_cast<float>(rand()) / RAND_MAX;
}

void run_dgemm(hipblasHandle_t handle, const double *dA, const double *dB, double *dC, double *dC2, int m, int n, int k) {
    const double alpha = 1.0, beta = 0.0;

    CHECK_CUBLAS(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, m, dB, k, &beta, dC, m));

    const unsigned num_moduli = 20u;                                    // Accuracy knob: 2 <= num_moduli <= 20
    const bool fastmode       = false;                                  // true (fast mode) or false (accurate mode)
    const size_t worksize     = gemmul8::workSize(m, n, k, num_moduli); // calculate required memory (Byte)
    void *work;
    hipMalloc(&work, worksize);
    gemmul8::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, m, dB, k, &beta, dC2, m, num_moduli, fastmode, work);

    std::vector<double> hC(m * n), hC2(m * n);
    CHECK_CUDA(hipMemcpy(hC.data(), dC, sizeof(double) * m * n, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(hC2.data(), dC2, sizeof(double) * m * n, hipMemcpyDeviceToHost));

    double err_chk = 0.;
    for (int i = 0; i < m * n; i++) {
        double diff = hC[i] - hC2[i];
        err_chk += diff * diff;
    }
    std::cout << "[DGEMM] L2 error: " << err_chk << std::endl;
    if (err_chk > 1e-3)
        std::cout << "L2 Error Too High!" << std::endl;

    hipFree(work);
}

void run_sgemm(hipblasHandle_t handle, const float *dA, const float *dB, float *dC, float *dC2, int m, int n, int k) {
    const float alpha = 1.0f, beta = 0.0f;

    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, m, dB, k, &beta, dC, m));

    const unsigned num_moduli = 20u;                                    // Accuracy knob: 2 <= num_moduli <= 20
    const bool fastmode       = false;                                  // true (fast mode) or false (accurate mode)
    const size_t worksize     = gemmul8::workSize(m, n, k, num_moduli); // calculate required memory (Byte)
    void *work;
    hipMalloc(&work, worksize);
    gemmul8::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA, m, dB, k, &beta, dC2, m, num_moduli, fastmode, work);

    std::vector<float> hC(m * n), hC2(m * n);
    CHECK_CUDA(hipMemcpy(hC.data(), dC, sizeof(float) * m * n, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(hC2.data(), dC2, sizeof(float) * m * n, hipMemcpyDeviceToHost));

    double err_chk = 0.;
    for (int i = 0; i < m * n; i++) {
        double diff = static_cast<double>(hC[i]) - static_cast<double>(hC2[i]);
        err_chk += diff * diff;
    }
    std::cout << "[SGEMM] L2 error: " << err_chk << std::endl;
    if (err_chk > 1e-3)
        std::cout << "L2 Error Too High!" << std::endl;

    hipFree(work);
}

int main() {
    srand(0);
    CHECK_CUDA(hipSetDevice(0));

    int m1 = 100, k1 = 80, n1 = 90;  // A1: m1×k1, B1: k1×n1
    int m2 = 120, k2 = 70, n2 = 110; // A2: m2×k2, B2: k2×n2

    double *A1, *A2, *B1, *B2;
    A1 = (double *)malloc(sizeof(double) * m1 * k1);
    A2 = (double *)malloc(sizeof(double) * m2 * k2);
    B1 = (double *)malloc(sizeof(double) * k1 * n1);
    B2 = (double *)malloc(sizeof(double) * k2 * n2);
    fill_random(A1, m1 * k1);
    fill_random(A2, m2 * k2);
    fill_random(B1, k1 * n1);
    fill_random(B2, k2 * n2);

    float *sA1, *sA2, *sB1, *sB2;
    sA1 = (float *)malloc(sizeof(float) * m1 * k1);
    sA2 = (float *)malloc(sizeof(float) * m2 * k2);
    sB1 = (float *)malloc(sizeof(float) * k1 * n1);
    sB2 = (float *)malloc(sizeof(float) * k2 * n2);
    fill_random(sA1, m1 * k1);
    fill_random(sA2, m2 * k2);
    fill_random(sB1, k1 * n1);
    fill_random(sB2, k2 * n2);

    double *dA1, *dA2, *dB1, *dB2, *dCd, *dCd2;
    float *dsA1, *dsA2, *dsB1, *dsB2, *dsC, *dsC2;

    CHECK_CUDA(hipMalloc(&dA1, sizeof(double) * m1 * k1));
    CHECK_CUDA(hipMalloc(&dA2, sizeof(double) * m2 * k2));
    CHECK_CUDA(hipMalloc(&dB1, sizeof(double) * k1 * n1));
    CHECK_CUDA(hipMalloc(&dB2, sizeof(double) * k2 * n2));

    CHECK_CUDA(hipMalloc(&dsA1, sizeof(float) * m1 * k1));
    CHECK_CUDA(hipMalloc(&dsA2, sizeof(float) * m2 * k2));
    CHECK_CUDA(hipMalloc(&dsB1, sizeof(float) * k1 * n1));
    CHECK_CUDA(hipMalloc(&dsB2, sizeof(float) * k2 * n2));

    int maxCd = std::max({m1 * n1, m1 * n2, m2 * n1, m2 * n2});
    int maxCs = maxCd;
    CHECK_CUDA(hipMalloc(&dCd, sizeof(double) * maxCd));
    CHECK_CUDA(hipMalloc(&dsC, sizeof(float) * maxCs));
    CHECK_CUDA(hipMalloc(&dCd2, sizeof(double) * maxCd));
    CHECK_CUDA(hipMalloc(&dsC2, sizeof(float) * maxCs));

    CHECK_CUDA(hipMemcpy(dA1, A1, sizeof(double) * m1 * k1, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA2, A2, sizeof(double) * m2 * k2, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB1, B1, sizeof(double) * k1 * n1, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB2, B2, sizeof(double) * k2 * n2, hipMemcpyHostToDevice));

    CHECK_CUDA(hipMemcpy(dsA1, sA1, sizeof(float) * m1 * k1, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dsA2, sA2, sizeof(float) * m2 * k2, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dsB1, sB1, sizeof(float) * k1 * n1, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dsB2, sB2, sizeof(float) * k2 * n2, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    std::cout << "Running GEMMs...\n";

    // ---- 2〜7: DGEMM ----
    run_dgemm(handle, dA1, dB1, dCd, dCd2, m1, n1, k1); // 2
    run_dgemm(handle, dA1, dB2, dCd, dCd2, m1, n2, k2); // 3
    run_dgemm(handle, dA1, dB1, dCd, dCd2, m1, n1, k1); // 4
    run_dgemm(handle, dA2, dB1, dCd, dCd2, m2, n1, k2); // 5
    run_dgemm(handle, dA2, dB1, dCd, dCd2, m2, n1, k2); // 6
    run_dgemm(handle, dA2, dB2, dCd, dCd2, m2, n2, k2); // 7

    // ---- 9〜14: SGEMM ----
    run_sgemm(handle, dsA1, dsB1, dsC, dsC2, m1, n1, k1); // 9
    run_sgemm(handle, dsA1, dsB2, dsC, dsC2, m1, n2, k2); // 10
    run_dgemm(handle, dA1, dB2, dCd, dCd2, m1, n2, k2);   // 11
    run_sgemm(handle, dsA2, dsB1, dsC, dsC2, m2, n1, k2); // 12
    run_sgemm(handle, dsA2, dsB1, dsC, dsC2, m2, n1, k2); // 13
    run_sgemm(handle, dsA1, dsB1, dsC, dsC2, m1, n1, k1); // 14

    std::cout << "All GEMMs completed.\n";

    CHECK_CUBLAS(hipblasDestroy(handle));

    hipFree(dA1);
    hipFree(dA2);
    hipFree(dB1);
    hipFree(dB2);
    hipFree(dsA1);
    hipFree(dsA2);
    hipFree(dsB1);
    hipFree(dsB2);
    hipFree(dCd);
    hipFree(dsC);
    hipFree(dCd2);
    hipFree(dsC2);

    free(A1);
    free(A2);
    free(B1);
    free(B2);
    free(sA1);
    free(sA2);
    free(sB1);
    free(sB2);

    return 0;
}
