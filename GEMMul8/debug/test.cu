#include "hip/hip_runtime.h"
#include "gemmul8.hpp"
#include "matrixmarketio.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>

void test(char trans_a, char trans_b, int moduli, bool fastmode) {

    std::cout << ((trans_a == 'N') ? "N" : "T") << ","
              << ((trans_b == 'N') ? "N" : "T") << ","
              << moduli << ","
              << int(fastmode) << ","
              << std::endl;

    Matrix<double> A;
    Matrix<double> B;
    Matrix<double> C;

    std::string matname = "problem";
    load_matrix("mats/" + matname + "_A.mtx", COORDINATE, A);
    load_matrix("mats/" + matname + "_B.mtx", COORDINATE, B);

    hipblasOperation_t TRANSA = (trans_a == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t TRANSB = (trans_b == 'N') ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasHandle_t cublasH   = NULL;
    hipStream_t stream      = NULL;
    const int M              = A.sz_rows;
    const int N              = B.sz_cols;
    const int K              = A.sz_cols;
    const double ALPHA       = 1.0;
    const double BETA        = 0.0;
    const int LDA            = (trans_a == 'N') ? M : K;
    const int LDB            = (trans_b == 'N') ? K : N;
    const int LDC            = M;
    const size_t LWORK       = gemmul8::workSize(M, N, K, moduli);

    const double *a = A.data;
    const double *b = B.data;
    double *dat_A   = nullptr;
    double *dat_B   = nullptr;
    double *dat_C   = nullptr;
    double *dat_Cg  = nullptr;
    void *WORK      = NULL;
    double *c       = new double[M * N];
    double *cg      = new double[M * N];

    hipblasCreate(&cublasH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(cublasH, stream);

    hipMalloc(&dat_A, sizeof(double) * M * K);
    hipMalloc(&dat_B, sizeof(double) * K * N);
    hipMalloc(&dat_C, sizeof(double) * M * N);

    hipMalloc(&dat_Cg, sizeof(double) * M * N);
    hipMalloc(&WORK, LWORK);

    hipMemcpy(dat_A, a, sizeof(double) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(dat_B, b, sizeof(double) * K * N, hipMemcpyHostToDevice);

    hipStreamSynchronize(stream);

    hipblasDgemm(cublasH, TRANSA, TRANSB, M, N, K, &ALPHA, dat_A, LDA, dat_B, LDB, &BETA, dat_C, LDC);

    hipStreamSynchronize(stream);

    gemmul8::gemm(cublasH, TRANSA, TRANSB, M, N, K, &ALPHA, dat_A, LDA, dat_B, LDB, &BETA, dat_Cg, LDC, moduli, fastmode, WORK);

    hipStreamSynchronize(stream);

    hipMemcpy(c, dat_C, sizeof(double) * M * N, hipMemcpyDeviceToHost);
    hipMemcpy(cg, dat_Cg, sizeof(double) * M * N, hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    hipFree(dat_A);
    hipFree(dat_B);
    hipFree(dat_C);
    hipFree(dat_Cg);
    hipFree(WORK);

    hipblasDestroy(cublasH);
    hipStreamDestroy(stream);

    double err_chk = 0.;
    for (int i = 0; i < M * N; i++) {
        err_chk += std::pow(c[i] - cg[i], 2);
    }

    std::cout << "(M,N,K): " << M << ", " << N << ", " << K << std::endl;
    std::cout << "L2: " << err_chk << std::endl;
    if (err_chk > 1e-4) {
        std::cout << "L2 Error Too High!" << std::endl;
    }

    delete[] c;
    delete[] cg;
}

int main() {
    test('N', 'N', 20, false);
    test('N', 'T', 20, false);
    test('T', 'N', 20, false);
    test('T', 'T', 20, false);

    test('N', 'N', 20, true);
    test('N', 'T', 20, true);
    test('T', 'N', 20, true);
    test('T', 'T', 20, true);
    return 0;
}
