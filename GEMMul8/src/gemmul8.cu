#include "hip/hip_runtime.h"
#include "../include/gemmul8.hpp"
#include "common.hpp"
#include "conv_32i_2_8u.hpp"
#include "inverse_scaling.hpp"
#include "scaling_accu.hpp"
#include "scaling_fast.hpp"

namespace gemmul8 {

//------------------------------
// Calculating required work size
//------------------------------
size_t workSize(const size_t m,            // Number of rows of C
                const size_t n,            // Number of columns of C
                const size_t k,            // Inner dimension <= 2^17
                const unsigned num_moduli) // #moduli, 2 <= num_moduli <= 20
{
    const size_t lda8i     = oz2::calc_ld8i(k);
    const size_t ldb8i     = lda8i;
    const size_t ldc32i    = oz2::calc_ld32i(m);
    const size_t sizeA     = lda8i * ldc32i;
    const size_t sizeB     = ldb8i * n;
    const size_t sizeC     = ldc32i * n;
    const size_t size_vecA = oz2::calc_sizevec(m);
    const size_t size_vecB = oz2::calc_sizevec(n);

    size_t total_size = 0;
    total_size += sizeof(int8_t) * (sizeA + sizeB) * num_moduli;
    total_size += sizeof(uint8_t) * sizeC * num_moduli;
    total_size += sizeof(int32_t) * sizeC;
    total_size += sizeof(int16_t) * (size_vecA + size_vecB);

    return total_size;
}

template <typename T>
__inline__ std::vector<double> gemm_(hipblasHandle_t handle,        // Handle to the cuBLAS library context
                                     const hipblasOperation_t op_A, // HIPBLAS_OP_N or HIPBLAS_OP_T
                                     const hipblasOperation_t op_B, // HIPBLAS_OP_N or HIPBLAS_OP_T
                                     const size_t m,               // Number of rows of C
                                     const size_t n,               // Number of columns of C
                                     const size_t k,               // Inner dimension <= 2^17
                                     const T *alpha,               // Scaling factor for op(A)*op(B)
                                     const T *const A,             // 1-D device array of dimensions lda*k (HIPBLAS_OP_N) or lda*m (HIPBLAS_OP_T)
                                     const size_t lda,             // Leading dimension of A
                                     const T *const B,             // 1-D device array of dimensions ldb*n (HIPBLAS_OP_N) or ldb*k (HIPBLAS_OP_T)
                                     const size_t ldb,             // Leading dimension of B
                                     const T *beta,                // Scaling factor for C
                                     T *const C,                   // 1-D device array of dimensions ldc*n
                                     const size_t ldc,             // Leading dimension of C
                                     const unsigned num_moduli,    // #moduli, 2 <= num_moduli <= 20
                                     const bool fastmode,          // false (accurate mode) or true (fast mode)
                                     void *const work)             // workspace allocated in advance
{
    //------------------------------
    // timer
    //------------------------------
    std::chrono::system_clock::time_point time_stamp;
    std::vector<double> timer(4, 0.0);

    //------------------------------
    // set constants
    //------------------------------
    const size_t lda8i       = oz2::calc_ld8i(k);
    const size_t ldb8i       = lda8i;
    const size_t ldc32i      = oz2::calc_ld32i(m);
    const size_t sizeA       = lda8i * ldc32i;
    const size_t sizeB       = ldb8i * n;
    const size_t sizeC       = ldc32i * n;
    const size_t size_vecA   = oz2::calc_sizevec(m);
    const unsigned table_idx = num_moduli - 2;
    constexpr int32_t one    = 1;
    constexpr int32_t zero   = 0;

    oz2::grid_invscal   = (m * n + oz2::threads_invscal - 1) / oz2::threads_invscal;
    oz2::grid_conv32i8u = ((sizeC >> 2) + oz2::threads_conv32i8u - 1) / oz2::threads_conv32i8u;

    bool is_numM_1;
    if constexpr (std::is_same_v<T, double>) {
        is_numM_1 = oz2_table::numM[table_idx] == 1;
        if (is_numM_1) {
            hipMemcpyToSymbol(HIP_SYMBOL(oz2_table::NMi_dev), &oz2_table::NMi_1[table_idx][0], num_moduli * sizeof(double));
        } else {
            hipMemcpyToSymbol(HIP_SYMBOL(oz2_table::NMi_dev), &oz2_table::NMi_2[num_moduli - 8][0][0], 2 * num_moduli * sizeof(double));
        }
        hipMemcpyToSymbol(HIP_SYMBOL(oz2_table::moduli_dev), oz2_table::moduli, num_moduli * sizeof(oz2_table::tab_t<double>));
    } else {
        is_numM_1 = true;
        hipMemcpyToSymbol(HIP_SYMBOL(oz2_table::NMi_dev), &oz2_table::NMi_1[table_idx][0], num_moduli * sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(oz2_table::modulif_dev), oz2_table::modulif, num_moduli * sizeof(oz2_table::tab_t<float>));
    }

    //------------------------------
    // set workspace (16byte align)
    //------------------------------
    int8_t *A8i   = reinterpret_cast<int8_t *>(work);                      // lda8i*m*sizeod(int8_t)*num_moduli
    int8_t *B8i   = A8i + sizeA * num_moduli;                              // ldb8i*n*sizeod(int8_t)*num_moduli
    uint8_t *C8u  = reinterpret_cast<uint8_t *>(B8i + sizeB * num_moduli); // (m*n+15)/16*16*sizeof(uint8_t)*num_moduli
    int32_t *C32i = reinterpret_cast<int32_t *>(C8u + sizeC * num_moduli); // (m*n+15)/16*16*sizeof(int32_t)
    int16_t *sftA = reinterpret_cast<int16_t *>(C32i + sizeC);             // (m+15)/16*16*sizeof(int16_t)
    int16_t *sftB = sftA + size_vecA;                                      // (n+15)/16*16*sizeof(int16_t)

    //------------------------------
    // Scaling
    // A =: diag(2^sftA) * A', A' is integer
    // B =: B' * diag(2^sftB), B' is integer
    // Then, calculating mod for all moduli
    // A8i := mod(A', modulus[i]) - 128 (-128 <= A8i <= 127)
    // B8i := mod(B', modulus[i]) - 128 (-128 <= A8i <= 127)
    //------------------------------
    oz2::timing(time_stamp);
    if (fastmode) {
        oz2::vecnorm::scaling<T>(op_A, op_B, m, n, k, num_moduli, A, lda, B, ldb, A8i, lda8i, sizeA, sftA, B8i, ldb8i, ldb8i * n, sftB, table_idx);
    } else {
        oz2::int8tc::scaling<T>(handle, op_A, op_B, m, n, k, num_moduli, A, lda, B, ldb, A8i, lda8i, sizeA, sftA, B8i, ldb8i, ldb8i * n, sftB, C32i, ldc32i, table_idx);
    }
    oz2::timing(time_stamp, timer[0]);

    for (unsigned i = 0; i < num_moduli; ++i) {
        //-----------------------------
        // Error-free matrix multiplication
        // C32i := A8i*B8i
        //------------------------------
        hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ldc32i, n, lda8i, &one, A8i + i * sizeA, HIP_R_8I, lda8i, B8i + i * sizeB, HIP_R_8I, ldb8i, &zero, C32i, HIP_R_32I, ldc32i, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
        oz2::timing(time_stamp, timer[1]);

        //------------------------------
        // Calculating mod
        // C8u[i] := mod(C32i, modulus[i]) >= 0
        //------------------------------
        oz2::conv_32i_2_8u(i, sizeC, C32i, C8u + i * sizeC);
        oz2::timing(time_stamp, timer[2]);
    }

    //------------------------------
    // Accumulation and Inverse scaling
    // C64f = sum(Ni*Mi*C8u[i]),
    //  where
    //      Mi := M/modulus[i],
    //      M := prod(modulus[all]),
    //      mod(Ni*Mi, modulus[i]) == 1.
    // C := C64f - round(C64f/M)*M
    // C := diag(2^-sftA) * C * diag(2^-sftB)
    //------------------------------
    oz2::inverse_scaling<T>(is_numM_1, num_moduli, m, n, C8u, ldc32i, sizeC, C, ldc, sftA, sftB, *alpha, *beta);
    oz2::timing(time_stamp, timer[3]);

    return timer;
}

template <> std::vector<double> gemm<double>(hipblasHandle_t handle,
                                             const hipblasOperation_t op_A,
                                             const hipblasOperation_t op_B,
                                             const size_t m,
                                             const size_t n,
                                             const size_t k,
                                             const double *alpha,
                                             const double *const A,
                                             const size_t lda,
                                             const double *const B,
                                             const size_t ldb,
                                             const double *beta,
                                             double *const C,
                                             const size_t ldc,
                                             const unsigned num_moduli,
                                             const bool fastmode,
                                             void *const work) //
{ return gemm_<double>(handle, op_A, op_B, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, num_moduli, fastmode, work); }
template <> std::vector<double> gemm<float>(hipblasHandle_t handle,
                                            const hipblasOperation_t op_A,
                                            const hipblasOperation_t op_B,
                                            const size_t m,
                                            const size_t n,
                                            const size_t k,
                                            const float *alpha,
                                            const float *const A,
                                            const size_t lda,
                                            const float *const B,
                                            const size_t ldb,
                                            const float *beta,
                                            float *const C,
                                            const size_t ldc,
                                            const unsigned num_moduli,
                                            const bool fastmode,
                                            void *const work) //
{ return gemm_<float>(handle, op_A, op_B, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc, num_moduli, fastmode, work); }

} // namespace gemmul8
